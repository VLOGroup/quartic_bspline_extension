#include "hip/hip_runtime.h"
#include <vector>
#include <torch/extension.h>

#include "include/constants.cuh"
#include "include/debug_utils.cuh"
#include "include/index_utils.cuh"
#include "include/device_utils.cuh"

/**
 * @brief CUDA kernel which performs evaluation of quartic (midpoint cardinal) b-spline potential function 
 *      and its derivative.
 * 
 * @note 
 *  > Assumption 1: Along every feature dimension, the same set of center nodes are used
 *  > Assumption 2: Center nodes are equally spaced in an interval [a, b]
 * 
 * @tparam T Floating point type: float, or double.
 * @param x Tensor of shape [bs, f, w, h] at which the b-spline has to be evaluated.
 * @param weight_tensor Tensor of shape [f, num_centers] corresponding to the weights spline potential
 *      at the center nodes for each marginal.
 * @param centers Tensor of shape [num_centers, ] of center nodes.
 * @param scale Scaling parameter.
 * @param scale_inv Inverse of the scaling parameter.
 * @param delta_inv Inverse of distance between (equally spaced) center nodes.
 * @param rho Tensor of shape [bs, f, w, h] - the evaluation of the spline potenital at x.
 * @param rho_prime Tensor of shape [bs, f, w, h] - the state-derivative of the spline potential 
 *      evaluated at x.
 */
template <typename T>
__global__ void quartic_bspline_forward_kernel(
    const torch::PackedTensorAccessor32<T, 4> x,
    const torch::PackedTensorAccessor32<T, 2> weight_tensor,
    const torch::PackedTensorAccessor32<T, 1> centers,
    const T scale,
    const T scale_inv,
    const T delta_inv,
    torch::PackedTensorAccessor32<T, 4> rho,
    torch::PackedTensorAccessor32<T, 4> rho_prime
){
    const int idx_h = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx_w = blockIdx.y * blockDim.y + threadIdx.y;
    const int idx_bf = blockIdx.z;

    const size_t num_features = x.size(1);
    const int idx_bs = idx_bf / num_features;
    const int idx_f = idx_bf % num_features;

    if (idx_bs < x.size(0) && idx_f < num_features && idx_w < x.size(2) && idx_h < x.size(3)){
        T rho_val = 0.0f;
        T rho_prime_val = 0.0f;
        const T x_ = x[idx_bs][idx_f][idx_w][idx_h];

        /**
         * Compute center index range corresponding to the current value of x - note that the
         * computation relies on the assumption of equally spaced center nodes!
         */
        const std::pair<size_t, size_t> center_idx_bounds = 
                    compute_center_index_bounds(x_, centers[0], scale, delta_inv, centers.size(0));

        for (size_t j = center_idx_bounds.first; j <= center_idx_bounds.second; j++){
            const T x_scaled = (x_ - centers[j]) * scale_inv;

            if (fabsf(x_scaled) < supp_rad){
                // determine support interval
                int interval = (int)(x_scaled - supp_lower);
                interval = max(0, min(num_supp_intervals - 1, interval));

                // evaluate local spline and its derivative
                T spline_val = quartic_bspline_coeffs[interval][4];
                T spline_deriv = 0.0f;

                #pragma unroll
                for (auto i = 1; i <= num_supp_intervals - 1; i++){
                    spline_deriv = spline_deriv * x_scaled + spline_val;
                    spline_val = spline_val * x_scaled 
                               + quartic_bspline_coeffs[interval][num_supp_intervals - 1 - i];
                }

                rho_val += weight_tensor[idx_f][j] * spline_val;
                rho_prime_val += weight_tensor[idx_f][j] * spline_deriv * scale_inv;
            }
        }
        rho[idx_bs][idx_f][idx_w][idx_h] = rho_val;
        rho_prime[idx_bs][idx_f][idx_w][idx_h] = rho_prime_val;
    }
}

std::vector<torch::Tensor> quartic_bspline_forward_function(
    const torch::Tensor x,
    const torch::Tensor weight_tensor,
    const torch::Tensor centers,
    const double scale
){
    check_device_and_datatype({x, weight_tensor, centers});

    const dim3 block_size(32, 8);
    const dim3 grid_size((x.size(3) + block_size.x - 1) / block_size.x, 
                         (x.size(2) + block_size.y - 1) / block_size.y,
                         x.size(0) * x.size(1));

    auto scalar_type = x.scalar_type();

    auto rho = torch::empty_like(x);
    auto rho_prime = torch::empty_like(x);

    const double scale_inv = 1.0 / scale;
    const double delta_inv = 1.0 / (centers[1].item<double>() - centers[0].item<double>());

    AT_DISPATCH_FLOATING_TYPES(scalar_type, "quartic_bspline_forward", [&] {
        quartic_bspline_forward_kernel<scalar_t><<<grid_size, block_size>>>(
            x.packed_accessor32<scalar_t, 4>(),
            weight_tensor.packed_accessor32<scalar_t, 2>(), 
            centers.packed_accessor32<scalar_t, 1>(),
            static_cast<scalar_t>(scale),
            static_cast<scalar_t>(scale_inv),
            static_cast<scalar_t>(delta_inv),
            rho.packed_accessor32<scalar_t, 4>(),
            rho_prime.packed_accessor32<scalar_t, 4>()
        );
    });

    CUDA_DEBUG_FUNC(hipGetLastError());

    return {rho, rho_prime};
}